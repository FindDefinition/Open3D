// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2020 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------
//

#define EIGEN_USE_GPU
#include "GroupPointsOpKernel.h"
#include "open3d/ml/Helper.h"
#include "open3d/ml/contrib/GroupPoints.cuh"
#include "open3d/ml/contrib/cuda_utils.h"

using namespace open3d;
using namespace open3d::ml;
using namespace open3d::ml::contrib;
using namespace tensorflow;

class GroupPointsOpKernelCUDA : public GroupPointsOpKernel {
public:
    explicit GroupPointsOpKernelCUDA(OpKernelConstruction *construction)
        : GroupPointsOpKernel(construction) {}

    void Kernel(tensorflow::OpKernelContext *context,
                int b,
                int c,
                int n,
                int npoints,
                int nsample,
                const float *points,
                const int *idx,
                float *out) {
        // points: (B, C, N)
        // idx: (B, npoints, nsample)
        // output:
        //      out: (B, C, npoints, nsample)

        auto stream = context->eigen_gpu_device().stream();

        hipError_t err;

        dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
                    b);  // blockIdx.x(col), blockIdx.y(row)
        dim3 threads(THREADS_PER_BLOCK);

        group_points_kernel<<<blocks, threads, 0, stream>>>(
                b, c, n, npoints, nsample, points, idx, out);
        // hipDeviceSynchronize();  // for using printf in kernel function
        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA kernel failed : %s\n",
                    hipGetErrorString(err));
            exit(-1);
        }
    }
};

REGISTER_KERNEL_BUILDER(Name("Open3DGroupPoints").Device(DEVICE_GPU),
                        GroupPointsOpKernelCUDA);

class GroupPointsGradOpKernelCUDA : public GroupPointsGradOpKernel {
public:
    explicit GroupPointsGradOpKernelCUDA(OpKernelConstruction *construction)
        : GroupPointsGradOpKernel(construction) {}

    void Kernel(tensorflow::OpKernelContext *context,
                int b,
                int c,
                int n,
                int npoints,
                int nsample,
                const float *grad_out,
                const int *idx,
                float *grad_points) {
        // grad_out: (B, C, npoints, nsample)
        // idx: (B, npoints, nsample)
        // output:
        //      grad_points: (B, C, N)

        auto stream = context->eigen_gpu_device().stream();

        hipError_t err;

        dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
                    b);  // blockIdx.x(col), blockIdx.y(row)
        dim3 threads(THREADS_PER_BLOCK);

        group_points_grad_kernel<<<blocks, threads, 0, stream>>>(
                b, c, n, npoints, nsample, grad_out, idx, grad_points);

        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA kernel failed : %s\n",
                    hipGetErrorString(err));
            exit(-1);
        }
    }
};

REGISTER_KERNEL_BUILDER(Name("Open3DGroupPointsGrad").Device(DEVICE_GPU),
                        GroupPointsGradOpKernelCUDA);