#include "hip/hip_runtime.h"
#include "open3d/ml/contrib/PointSampling.cuh"

namespace open3d {
namespace ml {
namespace contrib {

__global__ void gather_points_kernel(int b,
                                     int c,
                                     int n,
                                     int m,
                                     const float *__restrict__ points,
                                     const int *__restrict__ idx,
                                     float *__restrict__ out) {
    // points: (B, C, N)
    // idx: (B, M)
    // output:
    //      out: (B, C, M)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || c_idx >= c || pt_idx >= m) return;

    out += bs_idx * c * m + c_idx * m + pt_idx;
    idx += bs_idx * m + pt_idx;
    points += bs_idx * c * n + c_idx * n;
    out[0] = points[idx[0]];
}

__global__ void gather_points_grad_kernel(int b,
                                          int c,
                                          int n,
                                          int m,
                                          const float *__restrict__ grad_out,
                                          const int *__restrict__ idx,
                                          float *__restrict__ grad_points) {
    // grad_out: (B, C, M)
    // idx: (B, M)
    // output:
    //      grad_points: (B, C, N)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || c_idx >= c || pt_idx >= m) return;

    grad_out += bs_idx * c * m + c_idx * m + pt_idx;
    idx += bs_idx * m + pt_idx;
    grad_points += bs_idx * c * n + c_idx * n;

    atomicAdd(grad_points + idx[0], grad_out[0]);
}

}  // namespace contrib
}  // namespace ml
}  // namespace open3d
